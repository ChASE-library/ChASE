/* -*- Mode: C++; indent-tabs-mode: nil; c-basic-offset: 2 -*- */
// This file is a part of ChASE.
// Copyright (c) 2015-2023, Simulation and Data Laboratory Quantum Materials,
//   Forschungszentrum Juelich GmbH, Germany. All rights reserved.
// License is 3-clause BSD:
// https://github.com/ChASE-library/ChASE

#include "hipblas.h"
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>

#define BLOCKDIM 256
#define GRIDDIM 32

#define BLK_X 64
#define BLK_Y BLK_X

const int max_blocks = 65535;

static __device__ void dlacpy_full_device(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void dlacpy_full_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    dlacpy_full_device(m, n, dA, ldda, dB, lddb);
}


static __device__ void slacpy_full_device(
    int m, int n,
    const float *dA, int ldda,
    float       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void slacpy_full_kernel(
    int m, int n,
    const float *dA, int ldda,
    float       *dB, int lddb )
{
    slacpy_full_device(m, n, dA, ldda, dB, lddb);
}


static __device__ void zlacpy_full_device(
    int m, int n,
    const hipDoubleComplex *dA, int ldda,
    hipDoubleComplex       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__
void zlacpy_full_kernel(
    int m, int n,
    const hipDoubleComplex *dA, int ldda,
    hipDoubleComplex       *dB, int lddb )
{
    zlacpy_full_device(m, n, dA, ldda, dB, lddb);
}

static __device__ void clacpy_full_device(
    int m, int n,
    const hipComplex *dA, int ldda,
    hipComplex       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void clacpy_full_kernel(
    int m, int n,
    const hipComplex *dA, int ldda,
    hipComplex       *dB, int lddb )
{
    clacpy_full_device(m, n, dA, ldda, dB, lddb);
}


// generate `n` random float numbers on GPU
__global__ void s_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                float* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        v[i] = hiprand_normal(state);
    }
}

// generate `n` random double numbers on GPU
__global__ void d_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                double* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        v[i] = hiprand_normal_double(state);
    }
}
// generate `n` random complex single numbers on GPU
__global__ void c_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                hipComplex* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        float rnd = hiprand_normal(state);
        v[i].x = rnd;
        v[i].y = rnd;
    }
}

// generate `n` random complex double numbers on GPU
__global__ void z_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                hipDoubleComplex* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        double rnd = hiprand_normal_double(state);
        v[i].x = rnd;
        v[i].y = rnd;
    }
}

__global__ void sshift_matrix(float* A, int n, float shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx] += shift;
}

__global__ void dshift_matrix(double* A, int n, double shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx] += shift;
}

__global__ void cshift_matrix(hipComplex* A, int n, float shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx].x += shift;
}

__global__ void zshift_matrix(hipDoubleComplex* A, int n, double shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx].x += shift;
}

__global__ void sshift_mgpu_matrix(float* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, float shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
    }
}

__global__ void dshift_mgpu_matrix(double* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, double shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
    }
}

__global__ void cshift_mgpu_matrix(hipComplex* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, float shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind].x += shift;
    }
}

__global__ void zshift_mgpu_matrix(hipDoubleComplex* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, double shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind].x += shift;
    }
}
//only full copy is support right now
void t_lacpy_gpu(char uplo, int m, int n, float *dA, int ldda, float *dB, int lddb, hipStream_t stream_ )
{
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            slacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void t_lacpy_gpu(char uplo, int m, int n, double *dA, int ldda, double *dB, int lddb, hipStream_t stream_ )
{
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)	
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );     

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            dlacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}	

void t_lacpy_gpu(char uplo, int m, int n, std::complex<double> *ddA, int ldda, std::complex<double> *ddB, int lddb, hipStream_t stream_ )
{
    hipDoubleComplex *dA = reinterpret_cast<hipDoubleComplex*>(ddA);
    hipDoubleComplex *dB = reinterpret_cast<hipDoubleComplex*>(ddB);
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            zlacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void t_lacpy_gpu(char uplo, int m, int n, std::complex<float> *ddA, int ldda, std::complex<float> *ddB, int lddb, hipStream_t stream_ )
{
    hipComplex *dA = reinterpret_cast<hipComplex*>(ddA);
    hipComplex *dB = reinterpret_cast<hipComplex*>(ddB);
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            clacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states, float* v,
                       int n, hipStream_t stream_)
{
    s_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(seed, states, v, n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states, double* v,
                       int n, hipStream_t stream_)
{
    d_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(seed, states, v, n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                       std::complex<float>* v, int n, hipStream_t stream_)
{
    c_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(
        seed, states, reinterpret_cast<hipComplex*>(v), n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                       std::complex<double>* v, int n, hipStream_t stream_)
{
    z_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(
        seed, states, reinterpret_cast<hipDoubleComplex*>(v), n);
}

void chase_shift_matrix(float* A, int n, float shift, hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    sshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(A, n, shift);
}

void chase_shift_matrix(double* A, int n, double shift, hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    dshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(A, n, shift);
}

void chase_shift_matrix(std::complex<float>* A, int n, float shift,
                        hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    cshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
        reinterpret_cast<hipComplex*>(A), n, shift);
}

void chase_shift_matrix(std::complex<double>* A, int n, double shift,
                        hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    zshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
        reinterpret_cast<hipDoubleComplex*>(A), n, shift);
}

void chase_shift_mgpu_matrix(float* A, std::size_t* off_m, std::size_t* off_n,
                             std::size_t offsize, std::size_t ldH, float shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    sshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        A, off_m, off_n, offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(double* A, std::size_t* off_m, std::size_t* off_n,
                             std::size_t offsize, std::size_t ldH, double shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    dshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        A, off_m, off_n, offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(std::complex<float>* A, std::size_t* off_m,
                             std::size_t* off_n, std::size_t offsize,
                             std::size_t ldH, float shift, hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    cshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        reinterpret_cast<hipComplex*>(A), off_m, off_n,              //
        offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(std::complex<double>* A, std::size_t* off_m,
                             std::size_t* off_n, std::size_t offsize,
                             std::size_t ldH, double shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    zshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        reinterpret_cast<hipDoubleComplex*>(A), off_m, off_n,        //
        offsize, ldH, shift);
}
