/* -*- Mode: C++; indent-tabs-mode: nil; c-basic-offset: 2 -*- */
// This file is a part of ChASE.
// Copyright (c) 2015-2021, Simulation and Data Laboratory Quantum Materials, 
//   Forschungszentrum Juelich GmbH, Germany. All rights reserved.
// License is 3-clause BSD:
// https://github.com/ChASE-library/ChASE

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <complex>
#include "hipblas.h"

#define BLOCKDIM 256

__global__ void sshift_matrix(float* A, int n, float shift) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) A[(idx)*n + idx] += shift;
}

__global__ void dshift_matrix(double* A, int n, double shift) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) A[(idx)*n + idx] += shift;
}

__global__ void cshift_matrix(hipComplex* A, int n, float shift) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) A[(idx)*n + idx].x += shift;
}

__global__ void zshift_matrix(hipDoubleComplex* A, int n, double shift) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) A[(idx)*n + idx].x += shift;
}

__global__ void sshift_mgpu_matrix(float* A, std::size_t* off_m, std::size_t* off_n,
                                  std::size_t offsize, std::size_t ldH, float shift) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t ind;
  if(i < offsize){
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
  }
}

__global__ void dshift_mgpu_matrix(double* A, std::size_t* off_m, std::size_t* off_n,
                                  std::size_t offsize, std::size_t ldH, double shift) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t ind;
  if(i < offsize){
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
  }
}

__global__ void cshift_mgpu_matrix(hipComplex* A, std::size_t* off_m, std::size_t* off_n,
                                  std::size_t offsize, std::size_t ldH, float shift) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t ind;
  if(i < offsize){
        ind = off_n[i] * ldH + off_m[i];
        A[ind].x += shift;
  }
}

__global__ void zshift_mgpu_matrix(hipDoubleComplex* A, std::size_t* off_m, std::size_t* off_n,
                                  std::size_t offsize, std::size_t ldH, double shift) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t ind;
  if(i < offsize){
  	ind = off_n[i] * ldH + off_m[i];
	A[ind].x += shift;
  }
}

void chase_shift_matrix(float* A, int n, float shift,
                         hipStream_t* stream_) {
  int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
  sshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
      A, n, shift);
}

void chase_shift_matrix(double* A, int n, double shift,
                         hipStream_t* stream_) {
  int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
  dshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
      A, n, shift);
}

void chase_shift_matrix(std::complex<float>* A, int n, float shift,
                         hipStream_t* stream_) {
  int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
  cshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
      reinterpret_cast<hipComplex*>(A), n, shift);
}

void chase_shift_matrix(std::complex<double>* A, int n, double shift,
                         hipStream_t* stream_) {
  int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
  zshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
      reinterpret_cast<hipDoubleComplex*>(A), n, shift);
}

void chase_shift_mgpu_matrix(float* A, std::size_t* off_m, std::size_t* off_n,
                            std::size_t offsize, std::size_t ldH, float shift,
                             hipStream_t stream_) {

  unsigned int grid = (offsize + 256 - 1) / 256;
  dim3 threadsPerBlock(256, 1);
  dim3 numBlocks(grid, 1);
  sshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>(  //
      A, off_m, off_n, offsize, ldH, shift);

}


void chase_shift_mgpu_matrix(double* A, std::size_t* off_m, std::size_t* off_n,
                            std::size_t offsize, std::size_t ldH, double shift,
                             hipStream_t stream_) {

  unsigned int grid = (offsize + 256 - 1) / 256;
  dim3 threadsPerBlock(256, 1);
  dim3 numBlocks(grid, 1);
  dshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>(  //
      A, off_m, off_n, offsize, ldH, shift);

}

void chase_shift_mgpu_matrix(std::complex<float>* A, std::size_t* off_m, std::size_t* off_n,
                            std::size_t offsize, std::size_t ldH, float shift,
                             hipStream_t stream_) {

  unsigned int grid = (offsize + 256 - 1) / 256;
  dim3 threadsPerBlock(256, 1);
  dim3 numBlocks(grid, 1);
  cshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>(  //
      reinterpret_cast<hipComplex*>(A), off_m, off_n, //
                                offsize, ldH, shift);

}


void chase_shift_mgpu_matrix(std::complex<double>* A, std::size_t* off_m, std::size_t* off_n,
                            std::size_t offsize, std::size_t ldH, double shift,
                             hipStream_t stream_) {
  
  unsigned int grid = (offsize + 256 - 1) / 256;
  dim3 threadsPerBlock(256, 1);
  dim3 numBlocks(grid, 1);
  zshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>(  //
      reinterpret_cast<hipDoubleComplex*>(A), off_m, off_n, //
      				offsize, ldH, shift);

}

