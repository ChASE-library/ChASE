/* -*- Mode: C++; indent-tabs-mode: nil; c-basic-offset: 2 -*- */
// This file is a part of ChASE.
// Copyright (c) 2015-2023, Simulation and Data Laboratory Quantum Materials,
//   Forschungszentrum Juelich GmbH, Germany. All rights reserved.
// License is 3-clause BSD:
// https://github.com/ChASE-library/ChASE

#include "hipblas.h"
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>

#define BLOCKDIM 256
#define GRIDDIM 32

#define BLK_X 64
#define BLK_Y BLK_X
#define NB_X 64

const int max_blocks = 65535;

template< int n, typename T >
__device__ void cuda_sum_reduce(int i, T* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}

__global__ void c_resids_kernel(int m, int n, const hipComplex *A, int lda, const hipComplex *B, 
			 int ldb, float *ritzv, float *resids, bool is_sqrt )
{
    __shared__ float ssum[NB_X];
    int tx = threadIdx.x;
    A += blockIdx.x*lda;
    B += blockIdx.x*lda;
    
    ssum[tx] = 0;
    for(int i = tx; i < m; i += NB_X)
    {
        hipComplex alpha;
       	alpha.x = ritzv[blockIdx.x];
	alpha.y = 0.0;
	hipComplex a = hipCmulf(alpha, B[i]);
	hipComplex b = hipCsubf(A[i], a);
    	float nrm = hipCabsf(b);
	ssum[tx] += nrm * nrm;
    }

    cuda_sum_reduce<NB_X>(tx, ssum);
    if ( tx == 0 ) {
        if(is_sqrt)
	{
	    resids[ blockIdx.x ] = sqrtf(ssum[0]);	
	}
	else{
	    resids[ blockIdx.x ] = ssum[0];
    	}
    }
    
}

__global__ void z_resids_kernel(int m, int n, const hipDoubleComplex *A, int lda, const hipDoubleComplex *B,
                         int ldb, double *ritzv, double *resids, bool is_sqrt )
{
    __shared__ double ssum[NB_X];
    int tx = threadIdx.x;
    A += blockIdx.x*lda;
    B += blockIdx.x*lda;

    ssum[tx] = 0;
    for(int i = tx; i < m; i += NB_X)
    {
        hipDoubleComplex alpha;
        alpha.x = ritzv[blockIdx.x];
	alpha.y = 0.0;
        hipDoubleComplex a = hipCmul(alpha, B[i]);
        hipDoubleComplex b = hipCsub(A[i], a);
        double nrm = hipCabs(b);
	ssum[tx] += nrm * nrm;
    }

    cuda_sum_reduce<NB_X>(tx, ssum);
    if ( tx == 0 ) {
        if(is_sqrt)
        {
            resids[ blockIdx.x ] = sqrt(ssum[0]);
        }
        else{
            resids[ blockIdx.x ] = ssum[0];
        }
    }
}

__global__ void d_resids_kernel(int m, int n, const double *A, int lda, const double *B,
                         int ldb, double *ritzv, double *resids, bool is_sqrt )
{
    __shared__ double ssum[NB_X];
    int tx = threadIdx.x;
    A += blockIdx.x*lda;
    B += blockIdx.x*lda;

    ssum[tx] = 0;
    for(int i = tx; i < m; i += NB_X)
    {
        double alpha;
        alpha = ritzv[blockIdx.x];
        double a = alpha * B[i];
        double b = A[i] - a;
        ssum[tx] += b * b;
    }

    cuda_sum_reduce<NB_X>(tx, ssum);
    if ( tx == 0 ) {
        if(is_sqrt)
        {
            resids[ blockIdx.x ] = sqrt(ssum[0]);
        }
        else{
            resids[ blockIdx.x ] = ssum[0];
        }
    }
}

__global__ void s_resids_kernel(int m, int n, const float *A, int lda, const float *B,
                         int ldb, float *ritzv, float *resids, bool is_sqrt )
{
    __shared__ float ssum[NB_X];
    int tx = threadIdx.x;
    A += blockIdx.x*lda;
    B += blockIdx.x*lda;

    ssum[tx] = 0;
    for(int i = tx; i < m; i += NB_X)
    {
        float alpha;
        alpha = ritzv[blockIdx.x];
        float a = alpha * B[i];
        float b = A[i] - a;
        ssum[tx] += b * b;
    }

    cuda_sum_reduce<NB_X>(tx, ssum);
    if ( tx == 0 ) {
        if(is_sqrt)
        {
            resids[ blockIdx.x ] = sqrtf(ssum[0]);
        }
        else{
            resids[ blockIdx.x ] = ssum[0];
        }
    }
}

static __device__ void dlacpy_full_device(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void dlacpy_full_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    dlacpy_full_device(m, n, dA, ldda, dB, lddb);
}


static __device__ void slacpy_full_device(
    int m, int n,
    const float *dA, int ldda,
    float       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void slacpy_full_kernel(
    int m, int n,
    const float *dA, int ldda,
    float       *dB, int lddb )
{
    slacpy_full_device(m, n, dA, ldda, dB, lddb);
}


static __device__ void zlacpy_full_device(
    int m, int n,
    const hipDoubleComplex *dA, int ldda,
    hipDoubleComplex       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__
void zlacpy_full_kernel(
    int m, int n,
    const hipDoubleComplex *dA, int ldda,
    hipDoubleComplex       *dB, int lddb )
{
    zlacpy_full_device(m, n, dA, ldda, dB, lddb);
}

static __device__ void clacpy_full_device(
    int m, int n,
    const hipComplex *dA, int ldda,
    hipComplex       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    bool full = (iby + BLK_Y <= n);
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}

__global__ void clacpy_full_kernel(
    int m, int n,
    const hipComplex *dA, int ldda,
    hipComplex       *dB, int lddb )
{
    clacpy_full_device(m, n, dA, ldda, dB, lddb);
}


// generate `n` random float numbers on GPU
__global__ void s_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                float* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        v[i] = hiprand_normal(state);
    }
}

// generate `n` random double numbers on GPU
__global__ void d_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                double* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        v[i] = hiprand_normal_double(state);
    }
}
// generate `n` random complex single numbers on GPU
__global__ void c_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                hipComplex* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        float rnd = hiprand_normal(state);
        v[i].x = rnd;
        v[i].y = rnd;
    }
}

// generate `n` random complex double numbers on GPU
__global__ void z_normal_kernel(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                                hipDoubleComplex* v, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t* state = states + tid;
    hiprand_init(seed, tid, 0, state);

    int i;
    int nthreads = gridDim.x * blockDim.x;

    for (i = tid; i < n; i += nthreads)
    {
        double rnd = hiprand_normal_double(state);
        v[i].x = rnd;
        v[i].y = rnd;
    }
}

__global__ void sshift_matrix(float* A, int n, float shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx] += shift;
}

__global__ void dshift_matrix(double* A, int n, double shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx] += shift;
}

__global__ void cshift_matrix(hipComplex* A, int n, float shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx].x += shift;
}

__global__ void zshift_matrix(hipDoubleComplex* A, int n, double shift)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        A[(idx)*n + idx].x += shift;
}

__global__ void sshift_mgpu_matrix(float* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, float shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
    }
}

__global__ void dshift_mgpu_matrix(double* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, double shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind] += shift;
    }
}

__global__ void cshift_mgpu_matrix(hipComplex* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, float shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind].x += shift;
    }
}

__global__ void zshift_mgpu_matrix(hipDoubleComplex* A, std::size_t* off_m,
                                   std::size_t* off_n, std::size_t offsize,
                                   std::size_t ldH, double shift)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t ind;
    if (i < offsize)
    {
        ind = off_n[i] * ldH + off_m[i];
        A[ind].x += shift;
    }
}

void residual_gpu(int m, int n, std::complex<double> *dA, int lda, std::complex<double> *dB,
                         int ldb, double *d_ritzv, double *d_resids, bool is_sqrt, hipStream_t stream_)
{
    dim3 threads( NB_X);	
    dim3 grid( n );
    z_resids_kernel<<< grid, threads, 0, stream_ >>>( m, n, reinterpret_cast<hipDoubleComplex*>(dA), lda, reinterpret_cast<hipDoubleComplex*>(dB), ldb, d_ritzv, d_resids,is_sqrt);
}

void residual_gpu(int m, int n, std::complex<float> *dA, int lda, std::complex<float> *dB,
                         int ldb, float *d_ritzv, float *d_resids, bool is_sqrt, hipStream_t stream_)
{
    dim3 threads( NB_X);
    dim3 grid( n );
    c_resids_kernel<<< grid, threads, 0, stream_ >>>( m, n, reinterpret_cast<hipComplex*>(dA), lda, reinterpret_cast<hipComplex*>(dB), ldb, d_ritzv, d_resids,is_sqrt);
}

void residual_gpu(int m, int n, double *dA, int lda, double *dB,
                         int ldb, double *d_ritzv, double *d_resids, bool is_sqrt, hipStream_t stream_)
{
    dim3 threads( NB_X);
    dim3 grid( n );
    d_resids_kernel<<< grid, threads, 0, stream_ >>>( m, n, dA, lda, dB, ldb, d_ritzv, d_resids,is_sqrt);
}

void residual_gpu(int m, int n, float *dA, int lda, float *dB,
                         int ldb, float *d_ritzv, float *d_resids, bool is_sqrt, hipStream_t stream_)
{
    dim3 threads( NB_X);
    dim3 grid( n );
    s_resids_kernel<<< grid, threads, 0, stream_ >>>( m, n, dA, lda, dB, ldb, d_ritzv, d_resids,is_sqrt);
}

//only full copy is support right now
void t_lacpy_gpu(char uplo, int m, int n, float *dA, int ldda, float *dB, int lddb, hipStream_t stream_ )
{
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            slacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void t_lacpy_gpu(char uplo, int m, int n, double *dA, int ldda, double *dB, int lddb, hipStream_t stream_ )
{
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)	
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );     

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            dlacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}	

void t_lacpy_gpu(char uplo, int m, int n, std::complex<double> *ddA, int ldda, std::complex<double> *ddB, int lddb, hipStream_t stream_ )
{
    hipDoubleComplex *dA = reinterpret_cast<hipDoubleComplex*>(ddA);
    hipDoubleComplex *dB = reinterpret_cast<hipDoubleComplex*>(ddB);
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            zlacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void t_lacpy_gpu(char uplo, int m, int n, std::complex<float> *ddA, int ldda, std::complex<float> *ddB, int lddb, hipStream_t stream_ )
{
    hipComplex *dA = reinterpret_cast<hipComplex*>(ddA);
    hipComplex *dB = reinterpret_cast<hipComplex*>(ddB);
    #define dA(i_, j_) (dA + (i_) + (j_)*ldda)
    #define dB(i_, j_) (dB + (i_) + (j_)*lddb)
    int super_NB = max_blocks*BLK_X;
    dim3 super_grid(  (m + super_NB - 1) / super_NB,  (n + super_NB - 1) / super_NB );

    dim3 threads( BLK_X, 1 );
    dim3 grid;

    int mm, nn;
    for( unsigned int i=0; i < super_grid.x; ++i ) {
        mm = (i == super_grid.x-1 ? m % super_NB : super_NB);
        grid.x = ( mm + BLK_X - 1) / BLK_X;
        for( unsigned int j=0; j < super_grid.y; ++j ) {  // full row
            nn = (j == super_grid.y-1 ? n % super_NB : super_NB);
            grid.y = ( nn + BLK_X - 1) / BLK_Y;;
            clacpy_full_kernel <<< grid, threads, 0, stream_ >>>
                ( mm, nn, dA(i*super_NB, j*super_NB), ldda, dB(i*super_NB, j*super_NB), lddb );
        }
    }
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states, float* v,
                       int n, hipStream_t stream_)
{
    s_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(seed, states, v, n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states, double* v,
                       int n, hipStream_t stream_)
{
    d_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(seed, states, v, n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                       std::complex<float>* v, int n, hipStream_t stream_)
{
    c_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(
        seed, states, reinterpret_cast<hipComplex*>(v), n);
}

void chase_rand_normal(unsigned long long seed, hiprandStatePhilox4_32_10_t* states,
                       std::complex<double>* v, int n, hipStream_t stream_)
{
    z_normal_kernel<<<GRIDDIM, BLOCKDIM, 0, stream_>>>(
        seed, states, reinterpret_cast<hipDoubleComplex*>(v), n);
}

void chase_shift_matrix(float* A, int n, float shift, hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    sshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(A, n, shift);
}

void chase_shift_matrix(double* A, int n, double shift, hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    dshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(A, n, shift);
}

void chase_shift_matrix(std::complex<float>* A, int n, float shift,
                        hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    cshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
        reinterpret_cast<hipComplex*>(A), n, shift);
}

void chase_shift_matrix(std::complex<double>* A, int n, double shift,
                        hipStream_t* stream_)
{
    int num_blocks = (n + (BLOCKDIM - 1)) / BLOCKDIM;
    zshift_matrix<<<num_blocks, BLOCKDIM, 0, *stream_>>>(
        reinterpret_cast<hipDoubleComplex*>(A), n, shift);
}

void chase_shift_mgpu_matrix(float* A, std::size_t* off_m, std::size_t* off_n,
                             std::size_t offsize, std::size_t ldH, float shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    sshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        A, off_m, off_n, offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(double* A, std::size_t* off_m, std::size_t* off_n,
                             std::size_t offsize, std::size_t ldH, double shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    dshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        A, off_m, off_n, offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(std::complex<float>* A, std::size_t* off_m,
                             std::size_t* off_n, std::size_t offsize,
                             std::size_t ldH, float shift, hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    cshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        reinterpret_cast<hipComplex*>(A), off_m, off_n,              //
        offsize, ldH, shift);
}

void chase_shift_mgpu_matrix(std::complex<double>* A, std::size_t* off_m,
                             std::size_t* off_n, std::size_t offsize,
                             std::size_t ldH, double shift,
                             hipStream_t stream_)
{

    unsigned int grid = (offsize + 256 - 1) / 256;
    dim3 threadsPerBlock(256, 1);
    dim3 numBlocks(grid, 1);
    zshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
        reinterpret_cast<hipDoubleComplex*>(A), off_m, off_n,        //
        offsize, ldH, shift);
}
