#include "hip/hip_runtime.h"
#include "../include/filter.h"
#include <iostream>

#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include <omp.h>
#include <stdio.h>

#define BLOCKDIM 256

__global__ void zshift_matrix(hipDoubleComplex* A, int lda, int n, double shift, int offset)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if(idx  >= offset  && idx < n)
      A[(idx-offset) * lda + idx].x += shift;
}


static void handleError( hipError_t error, const char *file, int line ) {
    if (error != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( error ), file, line );
        exit( -1 );
    }
}


static void handleError_cublas( hipblasStatus_t error, const char *file, int line ) {
    if (error != HIPBLAS_STATUS_SUCCESS) {
       if(error == HIPBLAS_STATUS_INTERNAL_ERROR)
        printf( "HIPBLAS_STATUS_INTERNAL_ERROR in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_EXECUTION_FAILED)
        printf( "HIPBLAS_STATUS_EXECUTION_FAILED in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_MAPPING_ERROR)
        printf( "HIPBLAS_STATUS_MAPPING_ERROR in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_ARCH_MISMATCH)
        printf( "HIPBLAS_STATUS_ARCH_MISMATCH in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_INVALID_VALUE)
        printf( "HIPBLAS_STATUS_INVALID_VALUE in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_ALLOC_FAILED)
        printf( "HIPBLAS_STATUS_ALLOC_FAILED in %s at line %d\n", file, line );
       else if(error == HIPBLAS_STATUS_NOT_INITIALIZED)
        printf( "HIPBLAS_STATUS_NOT_INITIALIZED in %s at line %d\n", file, line );
       else
        printf( "Error code: %d in %s at line %d\n", error, file, line );
        exit( -1 );
    }
}

#define HANDLE_ERROR( error ) (handleError( (error) , __FILE__ , __LINE__ ))
#define HANDLE_ERROR_CUBLAS( error ) (handleError_cublas( (error) , __FILE__ , __LINE__ ))



int cuda_filter( MKL_Complex16 *H, MKL_Complex16 *V, int n, int unprocessed,
            int deg, int *degrees, double lambda_1, double lower, double upper,
            MKL_Complex16 *W )
{
  int N = n;
  int Av = 0;
  int num_mult = 0;

  hipDoubleComplex * H_d, *H_dO;
  hipDoubleComplex * V_d, *V_dO;
  hipDoubleComplex * W_d, *W_dO;
  MKL_Complex16 *V0, *W0;
  //hipDoubleComplex * tau_d;
//  double* ritzv_d;
  size_t sizeA, sizeV;

  hipblasHandle_t handle;
  hipStream_t stream;
  hipEvent_t event;

//  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleYield));
   //create stream
  hipSetDevice(0);
  hipStreamCreate(&stream);
  hipblasCreate(&handle);
  //all cublas tasks will be scheduled to this stream
  hipblasSetStream(handle, stream);

  //magma_init();

  sizeA = N * N * sizeof(hipDoubleComplex);
  sizeV = N * unprocessed * sizeof(hipDoubleComplex);

//   HANDLE_ERROR( hipMalloc((void**) &(ritzv_d), nevex * sizeof(double)) );  //sync
   HANDLE_ERROR( hipMalloc((void**) &(H_dO), sizeA) );  //sync
   HANDLE_ERROR( hipMalloc((void**) &(V_dO), sizeV) );  //sync
   HANDLE_ERROR( hipMalloc((void**) &(W_dO), sizeV) );  //sync
   hipEventCreate( &event );

   H_d = H_dO;
   V_d = V_dO;
   W_d = W_dO;
   W0 = W;
   V0 = V;

  hipDoubleComplex alpha, beta;
  double c = (upper + lower) / 2;
  double e = (upper - lower) / 2;
  double sigma_1   = e / (lambda_1 - c);
  double sigma     = sigma_1;
  double sigma_new;

    //---------------------------- y = alpha*(A-cI)*x --------------------------------
  alpha.x = sigma_1 / e;
  alpha.y = 0.0;
  beta.x = 0.0;
  beta.y = 0.0;

  //---------------------------- HtD transfer --------------------------------
  HANDLE_ERROR( hipMemcpyAsync(H_d, H, sizeA, hipMemcpyHostToDevice, stream)); //sync
  HANDLE_ERROR( hipMemcpyAsync(V_d, V, sizeV, hipMemcpyHostToDevice, stream)); //sync

  //----------------------------------- A = A-cI -------------------------------
  int num_blocks = (n+(BLOCKDIM-1))/BLOCKDIM;
  zshift_matrix<<<num_blocks, BLOCKDIM,0,stream>>>(H_d, n, n, -c, 0);


  HANDLE_ERROR_CUBLAS(
    hipblasZgemm(
      handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      n, unprocessed, n, &alpha,
      H_d, n, V_d, n, &beta, W_d, n
      )
    );

  Av += unprocessed;
  num_mult++;
  while( unprocessed >= 0 && *degrees <= num_mult )
  {
    degrees++; V+=n; W+=n; unprocessed--;
    V_d+=n; W_d+=n;
  };

  for( int i = 2; i <= deg; ++i )
    {
      sigma_new = 1.0 / ( 2.0/sigma_1 - sigma );

      //----------------------- V = alpha(A-cI)W + beta*V ----------------------
      alpha.x = 2.0*sigma_new / e;
      alpha.y = 0.0;
      beta.x  = -sigma * sigma_new;
      beta.y  = 0.0;

      HANDLE_ERROR_CUBLAS(
        hipblasZgemm(
          handle,
          HIPBLAS_OP_N, HIPBLAS_OP_N,
          n, unprocessed, n, &alpha,
          H_d, n, W_d, n, &beta, V_d, n
          )
        );

      sigma = sigma_new;

      std::swap( V, W );
      std::swap( V0, W0 );
      std::swap( V_d, W_d );
      std::swap( V_dO, W_dO );

      Av += unprocessed;
      num_mult++;
      while( unprocessed >= 0 && *degrees <= num_mult )
      {
        degrees++; V+=n; W+=n; unprocessed--;
        V_d+=n; W_d+=n;
      }

    } // for(i = 2; i <= deg; ++i)

  //---------------------------- DtH transfer --------------------------------
    HANDLE_ERROR(
      hipMemcpyAsync(V0, V_dO, sizeV, hipMemcpyDeviceToHost, stream) );
    HANDLE_ERROR(
      hipMemcpyAsync(W0, W_dO, sizeV, hipMemcpyDeviceToHost, stream) );

  //-----------------------------------RESTORE-A------------------------------------
  //zshift_matrix<<<num_blocks, BLOCKDIM, 0, stream>>>(H_d, n, n, c, 0);

    hipDeviceSynchronize();

   hipFree (H_dO);
   hipFree (V_dO);
   hipFree (W_dO);
   //magma_finalize();
   hipStreamDestroy(stream);
   hipblasDestroy(handle);


   return Av;
}
