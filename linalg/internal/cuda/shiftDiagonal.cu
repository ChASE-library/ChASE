#include "hip/hip_runtime.h"
// This file is a part of ChASE.
// Copyright (c) 2015-2024, Simulation and Data Laboratory Quantum Materials,
//   Forschungszentrum Juelich GmbH, Germany. All rights reserved.
// License is 3-clause BSD:
// https://github.com/ChASE-library/ChASE

#include "shiftDiagonal.cuh"

#define blockSize 256

namespace chase
{
namespace linalg
{
namespace internal
{
namespace cuda
{
    // ---------------------------------------- CUDA kernels ---------------------------------------------- //

    __global__ void sshift_matrix(float* A, std::size_t n, std::size_t lda, float shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] += shift;
    }
    __global__ void dshift_matrix(double* A, std::size_t n, std::size_t lda, double shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] += shift;

    }
    __global__ void cshift_matrix(hipComplex* A, std::size_t n, std::size_t lda, float shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx].x += shift;

    }
    __global__ void zshift_matrix(hipDoubleComplex* A, std::size_t n, std::size_t lda, double shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx].x += shift;

    }

    __global__ void sshift_mgpu_matrix(float* A, std::size_t* off_m,
                                    std::size_t* off_n, std::size_t offsize,
                                    std::size_t ldH, float shift)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        std::size_t ind;
        if (i < offsize)
        {
            ind = off_n[i] * ldH + off_m[i];
            A[ind] += shift;
        }
    }

    __global__ void dshift_mgpu_matrix(double* A, std::size_t* off_m,
                                    std::size_t* off_n, std::size_t offsize,
                                    std::size_t ldH, double shift)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        std::size_t ind;
        if (i < offsize)
        {
            ind = off_n[i] * ldH + off_m[i];
            A[ind] += shift;
        }
    }

    __global__ void cshift_mgpu_matrix(hipComplex* A, std::size_t* off_m,
                                    std::size_t* off_n, std::size_t offsize,
                                    std::size_t ldH, float shift)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        std::size_t ind;
        if (i < offsize)
        {
            ind = off_n[i] * ldH + off_m[i];
            A[ind].x += shift;
        }
    }

    __global__ void zshift_mgpu_matrix(hipDoubleComplex* A, std::size_t* off_m,
                                    std::size_t* off_n, std::size_t offsize,
                                    std::size_t ldH, double shift)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        std::size_t ind;
        if (i < offsize)
        {
            ind = off_n[i] * ldH + off_m[i];
            A[ind].x += shift;
        }
    }
    
    __global__ void ssubtract_inverse_diagonal(float* A, std::size_t n, std::size_t lda, float coef,
		    			       float* new_diag)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            new_diag[idx] = 1.0 / (coef - A[(idx) * lda + idx]);
    }
    __global__ void dsubtract_inverse_diagonal(double* A, std::size_t n, std::size_t lda, double coef,
		    			       double* new_diag)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            new_diag[idx] = 1.0 / (coef - A[(idx) * lda + idx]);
    }
    __global__ void csubtract_inverse_diagonal(hipComplex* A, std::size_t n, std::size_t lda, float coef,
		    			       float* new_diag)
    {
	//We assume the diagonal of A is real, and coef real. We quite new_diag complex for later operations
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            new_diag[idx] = 1.0 / (coef - A[(idx) * lda + idx].x);
    }
    __global__ void zsubtract_inverse_diagonal(hipDoubleComplex* A, std::size_t n, std::size_t lda, double coef,
		    			       double* new_diag)
    {
	//We assume the diagonal of A is real, and coef real. We quite new_diag complex for later operations
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            new_diag[idx] = 1.0 / (coef - A[(idx) * lda + idx].x);
    }
    
    __global__ void sset_diagonal(float* A, std::size_t n, std::size_t lda, float coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] = coef;
    }
    __global__ void dset_diagonal(double* A, std::size_t n, std::size_t lda, double coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] = coef;
    }
    __global__ void cset_diagonal(hipComplex* A, std::size_t n, std::size_t lda, hipComplex coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n){
            A[(idx) * lda + idx].x = coef.x;
            A[(idx) * lda + idx].y = coef.y;
	}
    }
    __global__ void zset_diagonal(hipDoubleComplex* A, std::size_t n, std::size_t lda, hipDoubleComplex coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n){
            A[(idx) * lda + idx].x = coef.x;
            A[(idx) * lda + idx].y = coef.y;
	}
    }
    __global__ void sscale_rows_matrix(float* A, std::size_t m, std::size_t n, std::size_t lda, float* coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	std::size_t coef_idx = idx % lda;
        if (coef_idx < m && idx < lda*n)
            A[idx] /= coef[coef_idx];
    }
    __global__ void dscale_rows_matrix(double* A, std::size_t m, std::size_t n, std::size_t lda, double* coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	std::size_t coef_idx = idx % lda;
        if (coef_idx < m && idx < lda*n)
            A[idx] /= coef[coef_idx];
    }
    __global__ void cscale_rows_matrix(hipComplex* A, std::size_t m, std::size_t n, std::size_t lda, float* coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	std::size_t coef_idx = idx % lda;
        if (coef_idx < m && idx < lda*n){
            A[idx].x /= coef[coef_idx];
            A[idx].y /= coef[coef_idx];
	}
    }
    __global__ void zscale_rows_matrix(hipDoubleComplex* A, std::size_t m, std::size_t n, std::size_t lda, double* coef)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	std::size_t coef_idx = idx % lda;
        if (coef_idx < m && idx < lda*n){
            A[idx].x /= coef[coef_idx];
            A[idx].y /= coef[coef_idx];
	}
    }

    // ------------------------------ ChASE templated calls to kernels ----------------------------------- //

    void chase_shift_matrix(float* A, std::size_t n, std::size_t lda, float shift, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        sshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, shift);
    }
    void chase_shift_matrix(double* A, std::size_t n, std::size_t lda,  double shift, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        dshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, shift);
    }
    void chase_shift_matrix(std::complex<float>* A, std::size_t n, std::size_t lda, float shift,
                            hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        cshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipComplex*>(A), n, lda, shift);

    }
    void chase_shift_matrix(std::complex<double>* A, std::size_t n, std::size_t lda, double shift,
                            hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        zshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipDoubleComplex*>(A), n, lda, shift);

    }

    void chase_shift_mgpu_matrix(float* A, std::size_t* off_m, std::size_t* off_n,
                                std::size_t offsize, std::size_t ldH, float shift,
                                hipStream_t stream_)
    {
        unsigned int grid = (offsize + blockSize - 1) / blockSize;
        if(grid == 0)
        {
            grid = 1;
        }
        dim3 threadsPerBlock(blockSize, 1);
        dim3 numBlocks(grid, 1);
        sshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
            A, off_m, off_n, offsize, ldH, shift);
    }

    void chase_shift_mgpu_matrix(double* A, std::size_t* off_m, std::size_t* off_n,
                                std::size_t offsize, std::size_t ldH, double shift,
                                hipStream_t stream_)
    {
        unsigned int grid = (offsize + blockSize - 1) / blockSize;
        if(grid == 0)
        {
            grid = 1;
        }
        dim3 threadsPerBlock(blockSize, 1);
        dim3 numBlocks(grid, 1);
        dshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
            A, off_m, off_n, offsize, ldH, shift);
    }

    void chase_shift_mgpu_matrix(std::complex<float>* A, std::size_t* off_m,
                                std::size_t* off_n, std::size_t offsize,
                                std::size_t ldH, float shift, hipStream_t stream_)
    {
        unsigned int grid = (offsize + blockSize - 1) / blockSize;
        if(grid == 0)
        {
            grid = 1;
        }
        dim3 threadsPerBlock(blockSize, 1);
        dim3 numBlocks(grid, 1);
        cshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
            reinterpret_cast<hipComplex*>(A), off_m, off_n,              //
            offsize, ldH, shift);
    }

    void chase_shift_mgpu_matrix(std::complex<double>* A, std::size_t* off_m,
                                std::size_t* off_n, std::size_t offsize,
                                std::size_t ldH, double shift,
                                hipStream_t stream_)
    {
        unsigned int grid = (offsize + blockSize - 1) / blockSize;
        if(grid == 0)
        {
            grid = 1;
        }
        dim3 threadsPerBlock(blockSize, 1);
        dim3 numBlocks(grid, 1);
        zshift_mgpu_matrix<<<numBlocks, threadsPerBlock, 0, stream_>>>( //
            reinterpret_cast<hipDoubleComplex*>(A), off_m, off_n,        //
            offsize, ldH, shift);
    }
    
    void chase_subtract_inverse_diagonal(float* A, std::size_t n, std::size_t lda, float coef,
		    			  float* new_diag, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        ssubtract_inverse_diagonal<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, coef, new_diag);
    }
    void chase_subtract_inverse_diagonal(double* A, std::size_t n, std::size_t lda, double coef,
		    			  double* new_diag, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        dsubtract_inverse_diagonal<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, coef, new_diag);
    }
    void chase_subtract_inverse_diagonal(std::complex<float>* A, std::size_t n, std::size_t lda, float coef,
		    			  float* new_diag, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        csubtract_inverse_diagonal<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipComplex*>(A), n, lda, coef,new_diag);
    }
    void chase_subtract_inverse_diagonal(std::complex<double>* A, std::size_t n, std::size_t lda, double coef,
		    			  double* new_diag, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        zsubtract_inverse_diagonal<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipDoubleComplex*>(A), n, lda, coef,new_diag);
    }
    
    void chase_set_diagonal(float* A, std::size_t n, std::size_t lda, float coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
	sset_diagonal<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, coef);
    }
    void chase_set_diagonal(double* A, std::size_t n, std::size_t lda, double coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        dset_diagonal<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, coef);
    }
    void chase_set_diagonal(std::complex<float>* A, std::size_t n, std::size_t lda, std::complex<float> coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        cset_diagonal<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipComplex*>(A), n, lda, make_hipComplex(std::real(coef),std::imag(coef)));
    }
    void chase_set_diagonal(std::complex<double>* A, std::size_t n, std::size_t lda, std::complex<double> coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        zset_diagonal<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipDoubleComplex*>(A), n, lda, make_hipDoubleComplex(std::real(coef),std::imag(coef)));
    }

    void chase_scale_rows_matrix(float* A, std::size_t m, std::size_t n, std::size_t lda, float* coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
	sscale_rows_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, m, n, lda, coef);
    }
    void chase_scale_rows_matrix(double* A, std::size_t m, std::size_t n, std::size_t lda, double* coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
	dscale_rows_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, m, n, lda, coef);
    }
    void chase_scale_rows_matrix(std::complex<float>* A, std::size_t m, std::size_t n, std::size_t lda, float* coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
	cscale_rows_matrix<<<num_blocks, blockSize, 0, stream_>>>(reinterpret_cast<hipComplex*>(A), m, n, lda, coef);
    } 
    void chase_scale_rows_matrix(std::complex<double>* A, std::size_t m, std::size_t n, std::size_t lda, double* coef, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
	zscale_rows_matrix<<<num_blocks, blockSize, 0, stream_>>>(reinterpret_cast<hipDoubleComplex*>(A), m, n, lda, coef);
    }
}
}
}
}
