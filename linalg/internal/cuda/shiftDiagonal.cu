#include "hip/hip_runtime.h"
#include "shiftDiagonal.cuh"

#define blockSize 256

namespace chase
{
namespace linalg
{
namespace internal
{
namespace cuda
{
    __global__ void sshift_matrix(float* A, std::size_t n, std::size_t lda, float shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] += shift;
    }
    __global__ void dshift_matrix(double* A, std::size_t n, std::size_t lda, double shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx] += shift;

    }
    __global__ void cshift_matrix(hipComplex* A, std::size_t n, std::size_t lda, float shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx].x += shift;

    }
    __global__ void zshift_matrix(hipDoubleComplex* A, std::size_t n, std::size_t lda, double shift)
    {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n)
            A[(idx) * lda + idx].x += shift;

    }

    void chase_shift_matrix(float* A, std::size_t n, std::size_t lda, float shift, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        sshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, shift);
    }
    void chase_shift_matrix(double* A, std::size_t n, std::size_t lda,  double shift, hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        dshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(A, n, lda, shift);
    }
    void chase_shift_matrix(std::complex<float>* A, std::size_t n, std::size_t lda, float shift,
                            hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        cshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipComplex*>(A), n, lda, shift);

    }
    void chase_shift_matrix(std::complex<double>* A, std::size_t n, std::size_t lda, double shift,
                            hipStream_t stream_)
    {
        std::size_t num_blocks = (n + (blockSize - 1)) / blockSize;
        zshift_matrix<<<num_blocks, blockSize, 0, stream_>>>(
            reinterpret_cast<hipDoubleComplex*>(A), n, lda, shift);

    }
}
}
}
}